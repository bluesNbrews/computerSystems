/*
**Author: Mark Williams
** Simple matrix multiplication using device code (NVIDIA GPU). 
** Matrix size is 16*16, values are all 32 and the output is 256.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width){

	//2D Thread ID
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//Pvalue stores the Pd element that is computed by the thread
	float Pvalue = 0;

	for(int k = 0; k < Width; ++k){
		float Mdelement = Md[ty * Width + k];
		float Ndelement = Nd[k * Width + tx];
		Pvalue += Mdelement * Ndelement; 
	}

	//Write the matrix to device memory each thread writes one element
	Pd[ty * Width + tx] = Pvalue;
}

void MatrixMultiplication(float* M, float* N, float* P, int Width){

	//Pointers for each matrix (separate matrices for device computation)
	float *Md, *Nd, *Pd;

	//Size to be used for memory allocation
	int size = Width * Width * sizeof(float);

	//Transfer matrix M and N to device memory
	hipMalloc((void**) &Md, size); 
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &Nd, size); 
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	
	//Allocate matrix P on the device
	hipMalloc((void**) &Pd, size); 

	//Setup the execution configuration
	dim3 dimBlock(Width, Width);
	dim3 dimGrid(1,1);

	//Launch the device computation threads
	MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);

	//Transfer P from device to host
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

	//Free device matrices
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
}


int main (){
	
	//Pointers for each matrix 
	float *M, *N, *P;
	
	//Use matrix width of 16, calculate size of memory for this width
	int Width = 16;
	int size = Width * Width * sizeof(float);
	int count = 0;

	//Allocate the memory
	M = (float*)malloc(size); 
	N = (float*)malloc(size);
	P = (float*)malloc(size);

	//Populate M and N with 1's and 2's, repsectively
	for(int i = 0; i < (Width * Width); i++){
		M[i] = 1.0;
		N[i] = 2.0;
	}

	//Pass the matrices and width size to multiplication function
	MatrixMultiplication(M, N, P, Width);

	//Print out the results
	for(int i = 0; i < (Width * Width); i++){
		count++;
		printf("%.0f ", P[i]);
		if(count % 16 == 0)
			printf("\n");
	}
	printf("Count is %d\n", count);

	//Deallocate memory
	free(M);
	free(N);
	free(P);

	return 0;
}
